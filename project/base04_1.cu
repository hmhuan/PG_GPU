#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/sort.h>

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

// histogram kernel
__global__ void computeHistKernel(uint32_t * in, int n, int * hist, int nBins, int bit)
{
    // TODO
    // Each block computes its local hist using atomic on SMEM
    //extern __shared__ int s_bin[];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    //int delta = (nBins - 1) / blockDim.x + 1;
    //for (int i = 0; i < delta; i++)
    //{
    //    int id = threadIdx.x + i * blockDim.x;
    //    if (id < nBins)
    //        s_bin[id] = 0;
    //}
    //__syncthreads();
    if (i < n)
    {
        int bin = (in[i] >> bit) & (nBins - 1);
        atomicAdd(&hist[bin], 1);
    }
    //__syncthreads();
    // Each block adds its local hist to global hist using atomic on GMEM
    //for (int i = 0; i < delta; i++)
    //{
    //    int id = threadIdx.x + i * blockDim.x;
    //    if (id < nBins)
    //        atomicAdd(&hist[id], s_bin[id]);
    //}

    
}

__global__ void scanBlkKernel(int * in, int n, int * out, int * blkSums)
{   
    // TODO
    extern __shared__ int s_data[];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > 0 && i < n)
       s_data[threadIdx.x] = in[i - 1];
    else
       s_data[threadIdx.x] = 0;
    __syncthreads();
    
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        int val = 0;
        if (threadIdx.x >= stride)
            val = s_data[threadIdx.x - stride];
        __syncthreads();
        s_data[threadIdx.x] += val;
        __syncthreads();
    }
    if (i < n)
        out[i] = s_data[threadIdx.x];
    if (blkSums != NULL)
        blkSums[blockIdx.x] = s_data[blockDim.x - 1];
}

void sortByHost(const uint32_t * in, int n,
                uint32_t * out,
                int nBits)
{
    int nBins = 1 << nBits; // 2^nBits
    int * hist = (int *)malloc(nBins * sizeof(int));
    int * histScan = (int *)malloc(nBins * sizeof(int));

    // In each counting sort, we sort data in "src" and write result to "dst"
    // Then, we swap these 2 pointers and go to the next counting sort
    // At first, we assign "src = in" and "dest = out"
    // However, the data pointed by "in" is read-only 
    // --> we create a copy of this data and assign "src" to the address of this copy
    uint32_t * src = (uint32_t *)malloc(n * sizeof(uint32_t));
    memcpy(src, in, n * sizeof(uint32_t));
    uint32_t * originalSrc = src; // Use originalSrc to free memory later
    uint32_t * dst = out;

    // Loop from LSD (Least Significant Digit) to MSD (Most Significant Digit)
    // (Each digit consists of nBits bits)
	// In each loop, sort elements according to the current digit 
	// (using STABLE counting sort)
    for (int bit = 0; bit < sizeof(uint32_t) * 8; bit += nBits)
    {
    	// TODO: Compute "hist" of the current digit
        memset(hist, 0, nBins * sizeof(int));
        for (int i = 0; i < n; i++)
        {
            int bin = (src[i] >> bit) & (nBins - 1);
            hist[bin]++;
        }

    	// TODO: Scan "hist" (exclusively) and save the result to "histScan"
        histScan[0] = 0;
        for (int bin = 1; bin < nBins; bin++)
            histScan[bin] = histScan[bin - 1] + hist[bin - 1];

    	// TODO: From "histScan", scatter elements in "src" to correct locations in "dst"
        for (int i = 0; i < n; i++)
        {
            int bin = (src[i] >> bit) & (nBins - 1);
            dst[histScan[bin]] = src[i];
            histScan[bin]++;
        }    

    	// TODO: Swap "src" and "dst"
        uint32_t * temp = src;
        src = dst;
        dst = temp; 
    }

    // TODO: Copy result to "out"
    memcpy(out, src, n * sizeof(uint32_t));
    // Free memories
    free(hist);
    free(histScan);
    free(originalSrc);
}

void sortRadixBase04(const uint32_t * in, int n, 
        uint32_t * out, 
        int nBits, int * blockSizes)
{

    dim3 blkSize1(blockSizes[0]); // block size for histogram kernel
    dim3 blkSize2(blockSizes[1]); // block size for scan kernel
    dim3 gridSize((n - 1) / blkSize1.x + 1); // grid size for histogram kernel 
    // TODO
    int nBins = 1 << nBits; // 2^nBits
    int * hist = (int *)malloc(nBins * gridSize.x * sizeof(int));
    int *histScan = (int * )malloc(nBins * gridSize.x * sizeof(int));


    uint32_t * src = (uint32_t *)malloc(n * sizeof(uint32_t));
    memcpy(src, in, n * sizeof(uint32_t));
    uint32_t * originalSrc = src; // Use originalSrc to free memory later
    uint32_t * dst = out;
    int * temp = (int *)malloc(nBins * gridSize.x * sizeof(int));

    for (int bit = 0; bit < sizeof(uint32_t) * 8; bit += nBits)
    {
    	// TODO: Compute "hist" of the current digit
        memset(hist, 0, nBins * gridSize.x * sizeof(int));
        for (int i = 0; i < gridSize.x; i++)
        {
            for (int j = 0; j < n / gridSize.x; j++)
            {
                int bin = (src[i * (n / gridSize.x) + j] >> bit) & (nBins - 1);
                hist[i * nBins + bin]++;
            }
        }
        int k = 1;
        histScan[0] = 0;
        temp[0] = 0;
        for (int j = 0; j < nBins; j++){
            for (int i = 0; i < gridSize.x; i++)
            {
                histScan[k] = histScan[k - 1] + hist[i * nBins + j];
                temp[j + i * nBins] = histScan[k - 1];
                k++;
            }
        }
        // // print hist scan
        // for (int i = 0; i < gridSize.x; i++)
        // {
        //     for (int j = 0; j < blkSize1.x; j++)
        //         printf("%d ", histScan[j * gridSize.x + i]);
        //     printf("\n");
        // }
        // printf("\n");
        // for (int i = 0; i < gridSize.x; i++)
        // {
        //     for (int j = 0; j < blkSize1.x; j++)
        //         printf("%d ", temp[i * blkSize1.x + j]);
        //     printf("\n");
        // }
        // printf("\n");

        for (int i = 0; i < gridSize.x; i++)
        {
            for (int j = 0; j < n / gridSize.x; j++)
            {
                int id = i * (n / gridSize.x) + j;
                int bin = i * nBins + (src[i * (n / gridSize.x) + j] >> bit) & (nBins - 1);
                //hist[i * blkSize1.x + bin]++;
                dst[temp[bin]] = src[id];
                temp[bin]++;  // (neu cung bin thi ghi ben canh)
            }
        }
        // TODO: Swap "src" and "dst"
        uint32_t * temp = src;
        src = dst;
        dst = temp; 
    }
    // TODO: Copy result to "out"
    memcpy(out, src, n * sizeof(uint32_t));
    // Free memories
    //free(blkSums);
    free(temp);
    free(hist);
    free(histScan);
    free(originalSrc);
}

void sort(const uint32_t * in, int n, 
        uint32_t * out, 
        int nBits,
        bool useDevice=false, int * blockSizes=NULL)
{
    GpuTimer timer; 
    timer.Start();

    if (useDevice == false)
    {
    	printf("\nRadix sort by host\n");
        sortByHost(in, n, out, nBits);
    }
    else 
    {
    	printf("\nRadix sort by device\n");
        sortRadixBase04(in, n, out, nBits, blockSizes);
    }

    timer.Stop();
    printf("Time: %.3f ms\n", timer.Elapsed());
}


void printDeviceInfo()
{
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
    printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
    printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
    printf("****************************\n");
}

void checkCorrectness(uint32_t * out, uint32_t * correctOut, int n)
{
    for (int i = 0; i < n; i++)
    {
        if (out[i] != correctOut[i])
        {
            printf("%d with %d != %d\n", i, out[i], correctOut[i]);
            printf("INCORRECT :(\n");
            return;
        }
    }
    printf("CORRECT :)\n");
}

void printArray(uint32_t * a, int n)
{
    for (int i = 0; i < n; i++)
        printf("%i ", a[i]);
    printf("\n");
}

int main(int argc, char ** argv)
{
	// PRINT OUT DEVICE INFO
    printDeviceInfo();

	// SET UP INPUT SIZE
    //uint32_t in[] = {1, 3, 5, 2, 2, 1, 6, 7, 3, 4, 4, 7}; // just for demo
    int nBits = 3;
    int n = (1 << 7) + 1;//sizeof(in) / sizeof(uint32_t); //;
    if (argc > 1)
        nBits = atoi(argv[1]);
    printf("\nInput size: %d\n", n);

    // ALLOCATE MEMORIES
    size_t bytes = n * sizeof(uint32_t);
    uint32_t * in = (uint32_t *)malloc(bytes);
    uint32_t * out = (uint32_t *)malloc(bytes); // Device result
    uint32_t * correctOut = (uint32_t *)malloc(bytes); // Host result

    // SET UP INPUT DATA
    for (int i = 0; i < n; i++)
       in[i] = rand() % 100 + 1;
    printf("n = %d\n", n);
   // printArray(in, n);
	
	// DETERMINE BLOCK SIZES
    int blockSizes[2] = {8, 8};//{512, 512}; // One for histogram, one for scan
    if (argc == 4)
    {
        blockSizes[0] = atoi(argv[2]);
        blockSizes[1] = atoi(argv[3]);
    }
    printf("\nHist block size: %d, scan block size: %d\n", blockSizes[0], blockSizes[1]);

    // SORT BY HOST
    sort(in, n, correctOut, nBits);
    //printArray(correctOut, n);
	sort(in, n, out, nBits, true, blockSizes);
    //printArray(out, n);
	checkCorrectness(out, correctOut, n);
    // FREE MEMORIES 
    free(in);
    free(out);
    free(correctOut);
    
    return EXIT_SUCCESS;
}