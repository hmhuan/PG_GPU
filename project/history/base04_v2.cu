#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/sort.h>

#define CHECK(call)                                                            \
{                                                                              \
    const hipError_t error = call;                                            \
    if (error != hipSuccess)                                                  \
    {                                                                          \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
        fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                hipGetErrorString(error));                                    \
        exit(1);                                                               \
    }                                                                          \
}

struct GpuTimer
{
    hipEvent_t start;
    hipEvent_t stop;

    GpuTimer()
    {
        hipEventCreate(&start);
        hipEventCreate(&stop);
    }

    ~GpuTimer()
    {
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }

    void Start()
    {
        hipEventRecord(start, 0);
        hipEventSynchronize(start);
    }

    void Stop()
    {
        hipEventRecord(stop, 0);
    }

    float Elapsed()
    {
        float elapsed;
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        return elapsed;
    }
};

void sortByHost(const uint32_t * in, int n,
                uint32_t * out,
                int nBits)
{
    int nBins = 1 << nBits; // 2^nBits
    int * hist = (int *)malloc(nBins * sizeof(int));
    int * histScan = (int *)malloc(nBins * sizeof(int));

    // In each counting sort, we sort data in "src" and write result to "dst"
    // Then, we swap these 2 pointers and go to the next counting sort
    // At first, we assign "src = in" and "dest = out"
    // However, the data pointed by "in" is read-only 
    // --> we create a copy of this data and assign "src" to the address of this copy
    uint32_t * src = (uint32_t *)malloc(n * sizeof(uint32_t));
    memcpy(src, in, n * sizeof(uint32_t));
    uint32_t * originalSrc = src; // Use originalSrc to free memory later
    uint32_t * dst = out;

    // Loop from LSD (Least Significant Digit) to MSD (Most Significant Digit)
    // (Each digit consists of nBits bits)
	// In each loop, sort elements according to the current digit 
	// (using STABLE counting sort)
    for (int bit = 0; bit < sizeof(uint32_t) * 8; bit += nBits)
    {
    	// TODO: Compute "hist" of the current digit
        memset(hist, 0, nBins * sizeof(int));
        for (int i = 0; i < n; i++)
        {
            int bin = (src[i] >> bit) & (nBins - 1);
            hist[bin]++;
        }   
    	// TODO: Scan "hist" (exclusively) and save the result to "histScan"
        histScan[0] = 0;
        for (int bin = 1; bin < nBins; bin++)
            histScan[bin] = histScan[bin - 1] + hist[bin - 1];

    	// TODO: From "histScan", scatter elements in "src" to correct locations in "dst"
        for (int i = 0; i < n; i++)
        {
            int bin = (src[i] >> bit) & (nBins - 1);
            dst[histScan[bin]] = src[i];
            histScan[bin]++;
        }    

    	// TODO: Swap "src" and "dst"
        uint32_t * temp = src;
        src = dst;
        dst = temp; 
    }

    // TODO: Copy result to "out"
    memcpy(out, src, n * sizeof(uint32_t));
    // Free memories
    free(hist);
    free(histScan);
    free(originalSrc);
}

void sortRadixBase04(const uint32_t * in, int n, uint32_t * out, int nBits, int * blockSizes)
{
    dim3 blkSize1(blockSizes[0]); // block size for histogram kernel
    dim3 blkSize2(blockSizes[1]); // block size for scan kernel
    dim3 gridSize((n - 1) / blkSize1.x + 1); // grid size for histogram kernel 
    // TODO
    int nBins = 1 << nBits; // 2^nBits
    int * hist = (int *)malloc(nBins * gridSize.x * sizeof(int));
    int * histScan = (int * )malloc(nBins * gridSize.x * sizeof(int));
    uint32_t * src = (uint32_t *)malloc(n * sizeof(uint32_t));
    memcpy(src, in, n * sizeof(uint32_t));
    uint32_t * originalSrc = src; // Use originalSrc to free memory later
    uint32_t * dst = out;

    int nHist = nBins * gridSize.x;
    for (int bit = 0; bit < sizeof(uint32_t) * 8; bit += nBits)
    {
        // TODO: Compute "hist" of the current digit
        memset(hist, 0, nBins * gridSize.x * sizeof(int));
        for (int i = 0; i < n; i++)
        {
            int bin = (src[i] >> bit) & (nBins - 1);
            hist[bin * gridSize.x + i / blkSize1.x]++;
        }
        // TODO: Exclusive scan
        histScan[0] = 0;
        for (int i = 1; i < nHist; i++)
            histScan[i] = histScan[i - 1] + hist[i - 1];
        
        // TODO: Scatter
        for (int i = 0; i < n ; i++)
        {
            int bin = i / blkSize1.x + ((src[i] >> bit) & (nBins - 1)) * gridSize.x;
            dst[histScan[bin]] = src[i];
            histScan[bin]++;
        }
        // TODO: Swap "src" and "dst"
        uint32_t * temp = src;
        src = dst;
        dst = temp; 
    }
    // TODO: Copy result to "out"
    memcpy(out, src, n * sizeof(uint32_t));
    // Free memories
    free(hist);
    free(histScan);
    free(originalSrc);
}

// histogram kernel
__global__ void computeHistKernel(uint32_t * in, int n, int * hist, int nBins, int bit)
{
    // TODO
    // Each block computes its local hist using atomic on SMEM
    // extern __shared__ int s_bin[];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    // int delta = (nBins - 1) / blockDim.x + 1;
    // for (int j = 0; j < delta; j++)
    // {
    //     int id = j * blockDim.x + threadIdx.x;
    //     if (id < nBins)
    //         s_bin[id] = 0;
    // }
    // __syncthreads();
    //int bin = -1;
    if (i < n)
    {
        int bin = (in[i] >> bit) & (nBins - 1);
        atomicAdd(&hist[bin * gridDim.x + blockIdx.x], 1);
        //atomicAdd(&s_bin[bin], 1);
    }
    // __syncthreads();
    // // Each block adds its local hist to global hist using atomic on GMEM
    // if (bin != -1)
    // {
    //     for (int j = 0; j < delta; j++)
    //     {
    //         int id = threadIdx.x + j * blockDim.x;
    //         if (id < nBins)
    //             atomicAdd(&hist[bin * gridDim.x + blockIdx.x], s_bin[bin]);
    //     }
    // }
}

__global__ void scanBlkKernel(int * in, int n, int * out, int * blkSums)
{   
    // TODO
    extern __shared__ int s_data[];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i > 0 && i < n)
        s_data[threadIdx.x] = in[i - 1];
    else
        s_data[threadIdx.x] = 0;
    __syncthreads();
    
    for (int stride = 1; stride < blockDim.x; stride *= 2)
    {
        int val = 0;
        if (threadIdx.x >= stride)
            val = s_data[threadIdx.x - stride];
        __syncthreads();
        s_data[threadIdx.x] += val;
        __syncthreads();
    }
    if (i < n)
        out[i] = s_data[threadIdx.x];
    if (blkSums != NULL)
        blkSums[blockIdx.x] = s_data[blockDim.x - 1];
}
__global__ void addBlkSums(int * in, int n, int* blkSums)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n && blockIdx.x > 0)
        in[i] += blkSums[blockIdx.x - 1];
}


__global__ void scatter(uint32_t * in, int bit, int *histScan, int n, int nBins, uint32_t *out)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
    {
        int bin = ((in[i] >> bit) & (nBins - 1));
        int val = histScan[blockIdx.x * nBins + bin];
        atomicAdd(&histScan[blockIdx.x * nBins + bin], 1);
        out[val] = in[i];
        //histScan[bin]++;  // (neu cung bin thi ghi ben canh)
        //atomicAdd(&histScan[blockIdx.x * nBins + bin], 1);
        // int bin = i * nBins + ((src[id] >> bit) & (nBins - 1));
        //             //hist[i * blkSize1.x + bin]++;
        //             dst[histScan[bin]] = src[id];
        //             histScan[bin]++;  // (neu cung bin thi ghi ben canh)
    }
}

void sortRadixBase04_device(const uint32_t * in, int n, 
        uint32_t * out, 
        int nBits, int * blockSizes)
{
    int nBins = 1 << nBits; // 2^nBits
    dim3 blkSize1(blockSizes[0]); // block size for histogram kernel
    dim3 blkSize2(blockSizes[1]); // block size for scan kernel
    dim3 gridSize1((n - 1) / blkSize1.x + 1); // grid size for histogram kernel 
    dim3 gridSize2((nBins * gridSize1.x - 1) / blkSize2.x + 1);
    // TODO
    int * hist = (int *)malloc(nBins * gridSize1.x * sizeof(int));
    int * scan = (int * )malloc(nBins * gridSize1.x * sizeof(int));
    int * blkSums = (int *)malloc(gridSize2.x * sizeof(int));

    uint32_t * src = (uint32_t *)malloc(n * sizeof(uint32_t));
    memcpy(src, in, n * sizeof(uint32_t));
    uint32_t * originalSrc = src; // Use originalSrc to free memory later
    uint32_t * dst = out;

    uint32_t * d_src;
    int *d_hist, *d_scan, *d_blkSums;

    CHECK(hipMalloc(&d_src, n * sizeof(uint32_t)));
	CHECK(hipMalloc(&d_hist, nBins * gridSize1.x * sizeof(int)));
	CHECK(hipMalloc(&d_scan, nBins * gridSize1.x * sizeof(int)));
	CHECK(hipMalloc(&d_blkSums, gridSize2.x * sizeof(int)));

    size_t sMemSize1 = nBins * sizeof(int); 
    size_t sMemSize2 = blkSize2.x * sizeof(int);
    
    for (int bit = 0; bit < sizeof(uint32_t) * 8; bit += nBits)
    {
    	// TODO: Compute "hist" of the current digit
        CHECK(hipMemcpy(d_src, src, n * sizeof(uint32_t), hipMemcpyHostToDevice));
        // TODO: Initialize d_hist using hipMemset
        CHECK(hipMemset(d_hist, 0, nBins * gridSize1.x * sizeof(int)));
        computeHistKernel<<<gridSize1, blkSize1, sMemSize1>>>(d_src, n, d_hist, nBins, bit);
        hipDeviceSynchronize();
	    CHECK(hipGetLastError());
        //CHECK(hipMemcpy(hist, d_hist, nBins * gridSize1.x * sizeof(int), hipMemcpyDeviceToHost));

        scanBlkKernel<<<gridSize2, blkSize2, sMemSize2>>>(d_hist, nBins * gridSize1.x, d_scan, d_blkSums);
        hipDeviceSynchronize();
	    CHECK(hipGetLastError());
        CHECK(hipMemcpy(blkSums, d_blkSums, gridSize2.x * sizeof(int), hipMemcpyDeviceToHost));
        for (int i = 1; i < gridSize2.x; i++)
            blkSums[i] += blkSums[i - 1];
        CHECK(hipMemcpy(d_blkSums, blkSums, gridSize2.x * sizeof(int), hipMemcpyHostToDevice));
        addBlkSums<<<gridSize2, blkSize2>>>(d_scan, nBins * gridSize1.x, d_blkSums);
        hipDeviceSynchronize();
	    CHECK(hipGetLastError());

        CHECK(hipMemcpy(scan, d_scan, nBins * gridSize1.x * sizeof(int), hipMemcpyDeviceToHost)); 
        
        // TODO: Scatter
        for (int i = 0; i < n ; i++)
        {
            int bin = i / blkSize1.x + ((src[i] >> bit) & (nBins - 1)) * gridSize1.x;
            dst[scan[bin]] = src[i];
            scan[bin]++;
        }
        // TODO: Swap "src" and "dst"
        uint32_t * temp = src;
        src = dst;
        dst = temp; 

    }
    // TODO: Copy result to "out"
    memcpy(out, src, n * sizeof(uint32_t));
    // Free memories
    CHECK(hipFree(d_src));
    CHECK(hipFree(d_hist));
    CHECK(hipFree(d_scan));
    CHECK(hipFree(d_blkSums));
    
    free(blkSums);
    free(hist);
    free(scan);
    free(originalSrc);
}

void sort(const uint32_t * in, int n, 
        uint32_t * out, 
        int nBits,
        int useDevice=0, int * blockSizes=NULL)
{
    GpuTimer timer; 
    timer.Start();

    if (useDevice == 0)
    {
    	printf("\nRadix sort by host\n");
        sortByHost(in, n, out, nBits);
    }
    else if (useDevice == 1)
    {
    	printf("\nRadix sort by  host level 1\n");
        sortRadixBase04(in, n, out, nBits, blockSizes);
    }
    else 
    {
        printf("\nRadix sort by device\n");
        sortRadixBase04_device(in, n, out, nBits, blockSizes);
    }
    timer.Stop();
    printf("Time: %.3f ms\n", timer.Elapsed());
}


void printDeviceInfo()
{
    hipDeviceProp_t devProv;
    CHECK(hipGetDeviceProperties(&devProv, 0));
    printf("**********GPU info**********\n");
    printf("Name: %s\n", devProv.name);
    printf("Compute capability: %d.%d\n", devProv.major, devProv.minor);
    printf("Num SMs: %d\n", devProv.multiProcessorCount);
    printf("Max num threads per SM: %d\n", devProv.maxThreadsPerMultiProcessor); 
    printf("Max num warps per SM: %d\n", devProv.maxThreadsPerMultiProcessor / devProv.warpSize);
    printf("GMEM: %zu byte\n", devProv.totalGlobalMem);
    printf("SMEM per SM: %zu byte\n", devProv.sharedMemPerMultiprocessor);
    printf("SMEM per block: %zu byte\n", devProv.sharedMemPerBlock);
    printf("****************************\n");
}

void checkCorrectness(uint32_t * out, uint32_t * correctOut, int n)
{
    for (int i = 0; i < n; i++)
    {
        if (out[i] != correctOut[i])
        {
            printf("%d with %d != %d\n", i, out[i], correctOut[i]);
            printf("INCORRECT :(\n");
            return;
        }
    }
    printf("CORRECT :)\n");
}

void printArray(uint32_t * a, int n)
{
    for (int i = 0; i < n; i++)
        printf("%i ", a[i]);
    printf("\n");
}

int main(int argc, char ** argv)
{
	// PRINT OUT DEVICE INFO
    printDeviceInfo();

	// SET UP INPUT SIZE
    //uint32_t in[] = {1, 3, 5, 2, 2, 1, 6, 7, 3, 4, 4, 7}; // just for demo
    int nBits = 8;
    int n = (1 << 24) + 1;//sizeof(in) / sizeof(uint32_t); //;
    if (argc > 1)
        nBits = atoi(argv[1]);
    printf("\nInput size: %d\n", n);
    printf("nBits: %d\n", nBits);

    // ALLOCATE MEMORIES
    size_t bytes = n * sizeof(uint32_t);
    uint32_t * in = (uint32_t *)malloc(bytes);
    uint32_t * out_0 = (uint32_t *)malloc(bytes); // base 4 host result
    uint32_t * out_1 = (uint32_t *)malloc(bytes); // base 4 Device result
    uint32_t * correctOut = (uint32_t *)malloc(bytes); // Host result

    // SET UP INPUT DATA
    for (int i = 0; i < n; i++)
       in[i] = rand() % 100;
	// DETERMINE BLOCK SIZES
    int blockSizes[2] = {512, 512}; // One for histogram, one for scan
    if (argc == 4)
    {
        blockSizes[0] = atoi(argv[2]);
        blockSizes[1] = atoi(argv[3]);
    }
    printf("\nHist block size: %d, scan block size: %d\n", blockSizes[0], blockSizes[1]);

    // SORT BY HOST
    sort(in, n, correctOut, nBits);
    //printArray(correctOut, n);
	sort(in, n, out_0, nBits, 1, blockSizes);
	checkCorrectness(out_0, correctOut, n);

	sort(in, n, out_1, nBits, 2, blockSizes);
	checkCorrectness(out_1, correctOut, n);
    //printArray(out_1, n);
    // FREE MEMORIES 
    free(in);
    free(out_0);
    free(out_1);
    free(correctOut);
    
    return EXIT_SUCCESS;
}