#include "hip/hip_runtime.h"
#include "myScatter.h"
#include <iostream>
using namespace std;

__global__ void myScatterKernel(uint32_t* in, uint32_t* scan, uint32_t* out, int n)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    if(i < n)
    {
        if(in[i] == 0)
        {
            out[i - scan[i]] = in[i];
            out[i - scan[i]] = 1;
        }
        // num0 = n - scan[n - 1] - in[n-1]
        else
        {
            //out[n - scan[n - 1] - in[n - 1] + scan[i]] = in[i];
            out[n - scan[n - 1] - in[n - 1] + scan[i]] = 1;
        }
    }
}

void MyScatter(const uint32_t* in, uint32_t *scan, uint32_t *out, int n, int blockSizeX)
{
    dim3 blockSize(blockSizeX), gridSize((n - 1)/blockSize.x + 1);

    uint32_t *d_in, *d_scan, *d_out;

    hipMalloc(&d_in, n*sizeof(uint32_t));
    hipMalloc(&d_scan, n*sizeof(uint32_t));
    hipMalloc(&d_out, n*sizeof(uint32_t));

    hipMemcpy(d_in, in, n*sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(d_scan, scan, n*sizeof(uint32_t), hipMemcpyHostToDevice);

    myScatterKernel<<<gridSize, blockSize>>>(d_in, d_scan, d_out, n);

    hipMemcpy(out, d_out, n*sizeof(uint32_t), hipMemcpyDeviceToHost);

    cout << "out ";
    for(int i = 0;i < n;i++)
        cout << out[i] << ' ';
    cout << endl;

    hipFree(d_scan), hipFree(d_out), hipFree(d_in);
}